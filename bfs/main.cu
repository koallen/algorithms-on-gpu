#include <hip/hip_runtime.h>
#include <iostream>

#include "bfs.cuh"

using namespace std;

int main()
{
	// test data
	int V[] = {0, 1, 2, 3, 5, 6, 7, 8, 9}; // the last one is not a vetex
	int E[] = {1, 3, 1, 2, 4, 5, 7, 4, 6};
	int C[] = {0, INF, INF, INF, INF, INF, INF, INF};

	cudaBfs(V, E, C, 8, 9, 0);

	cout << "Shortest distances are:" << endl;
	for (int i = 0; i < 8; ++i)
		cout << i << ": " << C[i] << endl;

	return 0;
}
