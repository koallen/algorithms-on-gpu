#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <cstdlib>
#include <cstring>
#include <cstdio>
#include <iostream>

#include "bfs.cuh"

#define BLOCK_SIZE 256

__global__ void cudaBfsKernel(int *, int *, int *, int *, int *, int);

int emptyFrontier(int *F, int vertexCount)
{
	for (int i = 0; i < vertexCount; ++i)
		if (F[i] == 1)
			return 0;
	return 1;
}

void cudaBfs(int *V, int *E, int *C, int vertexCount, int edgeCount, int source)
{
	// allocate frontier array
	int *F = (int *)malloc(sizeof(int) * vertexCount);
	memset(F, 0, sizeof(int) * vertexCount);

	// allocate visited array
	int *X = (int *)malloc(sizeof(int) * vertexCount);
	memset(X, 0, sizeof(int) * vertexCount);

	// update arrays for source
	F[source] = 1;

	// setup array on device
	int *V_d, *E_d, *F_d, *X_d, *C_d;
	hipMalloc(&V_d, sizeof(int) * (vertexCount + 1));
	hipMalloc(&E_d, sizeof(int) * edgeCount);
	hipMalloc(&F_d, sizeof(int) * vertexCount);
	hipMalloc(&X_d, sizeof(int) * vertexCount);
	hipMalloc(&C_d, sizeof(int) * vertexCount);
	hipMemcpy(V_d, V, sizeof(int) * (vertexCount + 1), hipMemcpyHostToDevice);
	hipMemcpy(E_d, E, sizeof(int) * edgeCount, hipMemcpyHostToDevice);
	hipMemcpy(F_d, F, sizeof(int) * vertexCount, hipMemcpyHostToDevice);
	hipMemcpy(X_d, X, sizeof(int) * vertexCount, hipMemcpyHostToDevice);
	hipMemcpy(C_d, C, sizeof(int) * vertexCount, hipMemcpyHostToDevice);

	// run the kernel
	dim3 grid((vertexCount+BLOCK_SIZE)/(BLOCK_SIZE), 1, 1);
	dim3 block(BLOCK_SIZE, 1, 1);
	do {
		cudaBfsKernel<<<grid, block>>>(V_d, E_d, F_d, X_d, C_d, vertexCount);
		hipDeviceSynchronize();
		hipMemcpy(F, F_d, sizeof(int) * vertexCount, hipMemcpyDeviceToHost);
	} while (!emptyFrontier(F, vertexCount));

	hipMemcpy(C, C_d, sizeof(int) * vertexCount, hipMemcpyDeviceToHost);

	hipFree(V_d);
	hipFree(E_d);
	hipFree(F_d);
	hipFree(X_d);
	hipFree(C_d);
}

__global__ void cudaBfsKernel(int *V, int *E, int *F, int *X, int *C, int VCount)
{
	int tid = blockDim.x * blockIdx.x + threadIdx.x;
	if (tid >= VCount) return;
	if (F[tid])
	{
		F[tid] = 0;
		X[tid] = 1;
		for (int i = V[tid]; i < V[tid+1]; ++i)
		{
			int nid = E[i];
			if (X[nid] != 1)
			{
				C[nid] = C[tid] + 1;
				F[nid] = 1;
			}
		}
	}
}
